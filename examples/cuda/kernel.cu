#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAddKernel(float* A, float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx than that on x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

// Wrapper function for the CUDA kernel
void cudaVecAdd(float* A, float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
